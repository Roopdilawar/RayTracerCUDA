#include "hip/hip_runtime.h"
// src/raytracer.cu

#include <hiprand/hiprand_kernel.h>
#include "Vector3.h"
#include "Ray.h"
#include "Camera.h"
#include "Sphere.h"
#include "HitRecord.h"
#include "Utils.h"

#include <hip/hip_runtime.h>
#include <>
#include <float.h>

#define MAX_DEPTH 50 // Adjust as needed

// Forward declaration of hit_world
__device__ bool hit_world(const Ray& r, float t_min, float t_max, HitRecord& rec, Sphere* spheres, int num_spheres);

// Initialize CURAND states kernel
__global__ void init_curand_kernel(hiprandState* states, int image_width, int image_height, unsigned long seed) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= image_width || j >= image_height) return;

    int idx = j * image_width + i;
    // Initialize CURAND state with a unique seed per thread
    hiprand_init(seed, idx, 0, &states[idx]);
}

// Iterative ray_color function with CURAND
__device__ Vector3 ray_color_iterative(Ray r, Sphere* spheres, int num_spheres, hiprandState* states, int pixel_index) {
    Vector3 color(1.0f, 1.0f, 1.0f); // Initialize color
    hiprandState local_state = states[pixel_index]; // Load CURAND state
    for (int depth = 0; depth < MAX_DEPTH; ++depth) {
        HitRecord rec;
        if (hit_world(r, 0.001f, FLT_MAX, rec, spheres, num_spheres)) {
            // Generate a random scatter direction using hemisphere sampling
            Vector3 scatter_direction = rec.normal + random_unit_vector(&local_state);

            // Update the ray
            r = Ray(rec.point, scatter_direction - rec.point);

            // Attenuation factor
            Vector3 attenuation(0.8f, 0.8f, 0.8f);
            color *= attenuation;

            // Update the state
            states[pixel_index] = local_state;
        } else {
            // Background gradient
            Vector3 unit_direction = r.direction.normalized();
            float t = 0.5f * (unit_direction.y + 1.0f);
            Vector3 bg_color = (1.0f - t) * Vector3(1.0f, 1.0f, 1.0f) + t * Vector3(0.5f, 0.7f, 1.0f);
            color *= bg_color;
            break; // Ray has left the scene
        }
    }
    return color;
}

__device__ bool hit_world(const Ray& r, float t_min, float t_max, HitRecord& rec, Sphere* spheres, int num_spheres) {
    HitRecord temp_rec;
    bool hit_anything = false;
    float closest_so_far = t_max;

    for (int i = 0; i < num_spheres; ++i) {
        if (spheres[i].hit(r, t_min, closest_so_far, temp_rec)) {
            hit_anything = true;
            closest_so_far = temp_rec.t;
            rec = temp_rec;
        }
    }
    return hit_anything;
}

__global__ void render_kernel(Vector3* framebuffer, int image_width, int image_height, Camera camera, Sphere* spheres, int num_spheres, hiprandState* states) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= image_width || j >= image_height) return;

    int pixel_index = j * image_width + i;

    float u = static_cast<float>(i) / (image_width - 1);
    float v = static_cast<float>(j) / (image_height - 1);
    Ray r = camera.get_ray(u, v);
    framebuffer[pixel_index] = ray_color_iterative(r, spheres, num_spheres, states, pixel_index);
}

// Host function to launch the ray tracing kernel
extern "C" void launch_raytracer(Vector3* framebuffer, int image_width, int image_height, Camera camera, Sphere* d_spheres, int num_spheres, hiprandState* d_states) {
    dim3 threads(16, 16);
    dim3 blocks((image_width + threads.x - 1) / threads.x,
                (image_height + threads.y - 1) / threads.y);

    render_kernel<<<blocks, threads>>>(framebuffer, image_width, image_height, camera, d_spheres, num_spheres, d_states);
    hipError_t err = hipGetLastError();


    hipDeviceSynchronize();
}

extern "C" void initialize_curand(hiprandState* d_states, int width, int height, unsigned long seed) {
    dim3 threads(16, 16);
    dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);

    // Launch kernel to initialize CURAND states
    init_curand_kernel<<<blocks, threads>>>(d_states, width, height, seed);
    hipDeviceSynchronize();
}